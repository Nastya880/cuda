
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>

__global__ void hello_kernel(void)
{
  printf("Hello, world from the device!\n");
}

int main(void)
{
  /*unsigned int timer;
  cutCreateTimer(&timer);
  curStartTimer(timer);*/

  printf("Hello, world from the host!\n");

  // инициализируем события
  hipEvent_t start, stop;
  float elapsedTime;

  // создаем события
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // запись события
  hipEventRecord(start, 0);
  hipEventRecord(stop,0);

  hello_kernel<<<1,1>>>(); 
 
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);

  // вывод информации 
  printf("Time spent executing by the GPU: %.7f  millseconds\n", elapsedTime);

  // уничтожение события 
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
