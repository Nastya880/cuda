#include "hip/hip_runtime.h"
%%writefile matrixMult.cu
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

__global__ void matrixMult(int *A, int *B, int *C, int N) 
{
    int i0 = N * (blockDim.y * blockIdx.y +  threadIdx.y);
    int j0 = blockDim.x * blockIdx.x + threadIdx.x;
    int sum = 0;

    for (int k = 0; k < N; k++)
    sum += A[i0 + k] * B[k * N + j0]; //смещение для записываемого элемента

    int ind = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    C[ind] = sum;
}

int main()
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // количество строк и столбцов матрицы
    int N = 100;
    
    size_t ABCsize = N * N * sizeof(int);
    
    int *h_A = (int *)malloc(ABCsize);
    int *h_B = (int *)malloc(ABCsize);
    int *h_C = (int *)malloc(ABCsize);
 
    //заполнение матриц
    for (int i = 0; i < N * N; ++i)
    {
        h_A[i] = rand() % (int)RAND_MAX;
    }

    for (int i = 0; i < N * N; ++i)
    {
        h_B[i] = rand()%(int)RAND_MAX;
    }
    
    int *d_A = NULL;
    hipMalloc((void **)&d_A, Asize);
    
    int *d_B = NULL;
    hipMalloc((void **)&d_B, Bsize);
    
    int * d_C = NULL;
    hipMalloc((void **)&d_C, Csize);
    hipMemcpy(d_A, h_A, ABCsize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, ABCsize, hipMemcpyHostToDevice);
 
    dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid = dim3(N / BLOCK_SIZE, N /  BLOCK_SIZE);
    hipEventRecord(start, 0);
    
    matrixMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", KernelTime);
    
    hipMemcpy(h_C, d_C, Csize, hipMemcpyDeviceToHost);
    
    printf("MULTIPLICATE\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            int sum = 0;
                for (int k = 0; k < N; k++) 
                    sum += h_A[i * N + k] * h_B[k * N + j];
        }
    }
    printf("END\n");
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
