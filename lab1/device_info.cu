#include <cstdio>
#include <hip/hip_runtime.h>
#include <chrono>
int main (int argc, char * argv [] )
{
    int deviceCount;
    hipDeviceProp_t devProp{};
    hipGetDeviceCount ( &deviceCount );
    printf ( "Found %d devices\n", deviceCount );
    for ( int device = 0; device < deviceCount; device++)
    {hipGetDeviceProperties ( &devProp, device );
        printf ("Device %d\n", device );
        printf ("Compute capability : %d.%d\n", devProp.major, devProp.minor);
        printf ("Name : %s\n", devProp.name);
        // Полный объем глобальной памяти в Mбайтах:
        printf ("Total Global Mem: %lu\n", (devProp.totalGlobalMem/(1024*1024)));
        printf ("Shared memory per block: %zu\n" , devProp.sharedMemPerBlock );
        printf ("Registers per block : %d\n", devProp.regsPerBlock);
        printf ("Warp size : %d\n", devProp.warpSize);
        printf ("Max threads per block: %d\n", devProp.maxThreadsPerBlock);
        printf ("Total constant memory: %zu\n", devProp.totalConstMem);
        printf ("Clock Rate : %d\n", devProp.clockRate);
        printf ("Texture Alignment : %zu\n", devProp.textureAlignment);
        printf ("Device Overlap : %d\n", devProp.deviceOverlap);
        printf ("Multiprocessor Count: %d\n", devProp.multiProcessorCount);
        printf ("Max Threads Dim : %d %d %d\n", devProp.maxThreadsDim[0],
                devProp.maxThreadsDim[1], devProp.maxThreadsDim[2] );
        printf ("Max Grid Size : %d %d %d\n", devProp.maxGridSize [0],
                devProp.maxGridSize [1], devProp.maxGridSize [2]);
        printf("");
    }
    return 0;
}
