#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

constexpr const size_t N = 1024 * 1024;

__global__ void kernel(float* data)
{
}

void run_hello_world(dim3 grid_dim, dim3 block_dim)
{
    float* a;
    float* dev = nullptr;

    a = (float*)malloc(N * sizeof(float));
    hipMalloc((void**)&dev, N * sizeof(float));

    auto start = std::chrono::high_resolution_clock::now();
    kernel<<<grid_dim, block_dim>>> (dev);
    auto end = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count() << "ns\n";

    hipMemcpy(a, dev, N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(dev);
    free(a);
}

int main(int argc, char* argv[])
{

    int deviceCount;

    std::cout << "Base GPU execution time: ";
    run_hello_world({(N/512),1}, {512, 1});

//    start = std::chrono::high_resolution_clock::now();
    std::cout << "Full GPU execution time: ";
    run_hello_world({(N/prop.maxThreadsDim[0]),1}, {(uint32_t)prop.maxThreadsDim[0], 1});
//    end = std::chrono::high_resolution_clock::now();
//    std::cout << "Full GPU execution time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "ms\n";

    return 0;
}
